#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <string>
#include <cstdint>
#include <vector>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

//stb image headers
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

//cuda kernel to detect edges within the given imagedata
__global__ void edge_detection_on_gpu( uint8_t *image_data, uint8_t *output_data, int image_width, int image_height, int component_count )
{
    int pixelX = threadIdx.x + blockIdx.x * 32;
    int pixelY = threadIdx.y + blockIdx.y * 32;

    //sobel edge detection
    const int convolution_matrix_1[3][3] = {
        {  1,  0, -1 },
        {  2,  0, -2 },
        {  1,  0, -1 }
    };

    const int convolution_matrix_2[3][3] = {
        {  1,  2,  1 },
        {  0,  0,  0 },
        { -1, -2, -1 }
    };

    //only execute if the current pixel x and y are within the boundery of the image
    if ( ( component_count == 4 ) && ( ( pixelX < image_width ) && ( pixelY < image_height ) ) )
    {
        if ( ( pixelX > 2 ) && ( pixelY > 2 ) )
        {
            int final_sum_1_r = 0;
            int final_sum_1_g = 0;
            int final_sum_1_b = 0;
            int final_sum_2_r = 0;
            int final_sum_2_g = 0;
            int final_sum_2_b = 0;

            //do the convolution
            for ( int y = pixelY; y > ( pixelY - 3 ); y-- )
            {
                for ( int x = pixelX; x > ( pixelX - 3 ); x-- )
                {
                    int convolution_value_1 = ( convolution_matrix_1[ -( x - pixelX ) ][ -( y - pixelY ) ] );
                    int convolution_value_2 = ( convolution_matrix_2[ -( x - pixelX ) ][ -( y - pixelY ) ] );
                    final_sum_1_r += ( ( image_data[ ( x + y * image_width ) * component_count + 0 ] ) * convolution_value_1 );
                    final_sum_1_g += ( ( image_data[ ( x + y * image_width ) * component_count + 1 ] ) * convolution_value_1 );
                    final_sum_1_b += ( ( image_data[ ( x + y * image_width ) * component_count + 2 ] ) * convolution_value_1 );
                    final_sum_2_r += ( ( image_data[ ( x + y * image_width ) * component_count + 0 ] ) * convolution_value_2 );
                    final_sum_2_g += ( ( image_data[ ( x + y * image_width ) * component_count + 1 ] ) * convolution_value_2 );
                    final_sum_2_b += ( ( image_data[ ( x + y * image_width ) * component_count + 2 ] ) * convolution_value_2 );
                }
            }

            //combine the two convolutions
            uint8_t final_sum_r = ( uint8_t )( ( int )sqrtf( final_sum_1_r * final_sum_1_r + final_sum_2_r * final_sum_2_r ) & 0xFF );
            uint8_t final_sum_g = ( uint8_t )( ( int )sqrtf( final_sum_1_g * final_sum_1_g + final_sum_2_g * final_sum_2_g ) & 0xFF );
            uint8_t final_sum_b = ( uint8_t )( ( int )sqrtf( final_sum_1_b * final_sum_1_b + final_sum_2_b * final_sum_2_b ) & 0xFF );

            //write data to the correct memory address for the new image ( -3 pixels )
            output_data[ ( ( pixelX - 3 ) + ( pixelY - 3 ) * ( image_width - 3 ) ) * component_count + 0 ] = final_sum_r;
            output_data[ ( ( pixelX - 3 ) + ( pixelY - 3 ) * ( image_width - 3 ) ) * component_count + 1 ] = final_sum_g;
            output_data[ ( ( pixelX - 3 ) + ( pixelY - 3 ) * ( image_width - 3 ) ) * component_count + 2 ] = final_sum_b;
            output_data[ ( ( pixelX - 3 ) + ( pixelY - 3 ) * ( image_width - 3 ) ) * component_count + 3 ] = 255;
        }
    }
}

//cuda kernel to convert image to grayscale
__global__ void gray_scale_on_gpu( uint8_t *image_data, uint8_t *output_data, int image_width, int image_height, int component_count )
{
    int pixelX = threadIdx.x + blockIdx.x * 32;
    int pixelY = threadIdx.y + blockIdx.y * 32;

    //only execute if the current pixel x and y are within the boundery of the image
    if ( ( component_count == 4 ) && ( ( pixelX < image_width ) && ( pixelY < image_height ) ) )
    {
        //grayscale formula
        uint8_t pixel_value = ( uint8_t )( image_data[ ( pixelX + pixelY * image_width ) * component_count + 0 ] * 0.2126f + image_data[ ( pixelX + pixelY * image_width ) * component_count + 1 ] * 0.7152f + image_data[ ( pixelX + pixelY * image_width ) * component_count + 2 ] * 0.0722f);
        
        //write data to the correct memory address for the new image
        output_data[ ( pixelX + pixelY * image_width ) * component_count + 0 ] = pixel_value;
        output_data[ ( pixelX + pixelY * image_width ) * component_count + 1 ] = pixel_value;
        output_data[ ( pixelX + pixelY * image_width ) * component_count + 2 ] = pixel_value;
        output_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ] = image_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ];
    }
}

//cuda kernel to do average pooling on image data
__global__ void average_pooling_on_gpu( uint8_t *image_data, uint8_t *output_data, int image_width, int image_height, int component_count )
{
    int pixelX = threadIdx.x + blockIdx.x * 32;
    int pixelY = threadIdx.y + blockIdx.y * 32;

    //only execute if the current pixel x and y are within the boundery of the image
    if ( ( component_count == 4 ) && ( ( pixelX < image_width ) && ( pixelY < image_height ) ) )
    {
        //stride to a 2x2 grid
        if ( ( ( ( pixelX - 1 ) % 2 == 0 ) && ( ( pixelY - 1 ) % 2 == 0 ) ) )
        {
            int final_sum_r = 0;
            int final_sum_g = 0;
            int final_sum_b = 0;

            //get the sum of all values in a 2x2 square
            for ( int y = pixelY; y > ( pixelY - 2 ); y-- )
            {
                for ( int x = pixelX; x > ( pixelX - 2 ); x-- )
                {
                    final_sum_r += image_data[ ( x + y * image_width ) * component_count + 0 ];
                    final_sum_g += image_data[ ( x + y * image_width ) * component_count + 1 ];
                    final_sum_b += image_data[ ( x + y * image_width ) * component_count + 2 ];
                }
            }

            //write data to the correct memory address for the new image ( /2 )
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 0 ] = final_sum_r / 4; // sum devided by 4 for average formula
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 1 ] = final_sum_g / 4;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 2 ] = final_sum_b / 4;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 3 ] = image_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ];
        }
    }
}

//cuda kernel for doing max pooling on image data
__global__ void max_pooling_on_gpu( uint8_t *image_data, uint8_t *output_data, int image_width, int image_height, int component_count )
{
    int pixelX = threadIdx.x + blockIdx.x * 32;
    int pixelY = threadIdx.y + blockIdx.y * 32;

    //only execute if the current pixel x and y are within the boundery of the image
    if ( ( component_count == 4 ) && ( ( pixelX < image_width ) && ( pixelY < image_height ) ) )
    {
        //stride to a 2x2 grid
        if ( ( ( ( pixelX - 1 ) % 2 == 0 ) && ( ( pixelY - 1 ) % 2 == 0 ) ) )
        {
            int final_max_r = 0;
            int final_max_g = 0;
            int final_max_b = 0;

            //find max value in a 2x2 grid
            for ( int y = pixelY; y > ( pixelY - 2 ); y-- )
            {
                for ( int x = pixelX; x > ( pixelX - 2 ); x-- )
                {
                    final_max_r = final_max_r > image_data[ ( x + y * image_width ) * component_count + 0 ] ? final_max_r : image_data[ ( x + y * image_width ) * component_count + 0 ];
                    final_max_g = final_max_g > image_data[ ( x + y * image_width ) * component_count + 1 ] ? final_max_g : image_data[ ( x + y * image_width ) * component_count + 1 ];
                    final_max_b = final_max_b > image_data[ ( x + y * image_width ) * component_count + 2 ] ? final_max_b : image_data[ ( x + y * image_width ) * component_count + 2 ];
                }
            }

            //write data to the correct memory address for the new image ( /2 )
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 0 ] = final_max_r;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 1 ] = final_max_g;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 2 ] = final_max_b;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 3 ] = image_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ];
        }
    }
}

//cuda kernel for doing minimum pooling on image data
__global__ void min_pooling_on_gpu( uint8_t *image_data, uint8_t *output_data, int image_width, int image_height, int component_count )
{
    int pixelX = threadIdx.x + blockIdx.x * 32;
    int pixelY = threadIdx.y + blockIdx.y * 32;

    //only execute if the current pixel x and y are within the boundery of the image
    if ( ( component_count == 4 ) && ( ( pixelX < image_width ) && ( pixelY < image_height ) ) )
    {
        //stride to a 2x2 grid
        if ( ( ( ( pixelX - 1 ) % 2 == 0 ) && ( ( pixelY - 1 ) % 2 == 0 ) ) )
        {
            int final_min_r = 255;
            int final_min_g = 255;
            int final_min_b = 255;

            //find the lowest value starting from 255 ( highest )
            for ( int y = pixelY; y > ( pixelY - 2 ); y-- )
            {
                for ( int x = pixelX; x > ( pixelX - 2 ); x-- )
                {
                    final_min_r = final_min_r < image_data[ ( x + y * image_width ) * component_count + 0 ] ? final_min_r : image_data[ ( x + y * image_width ) * component_count + 0 ];
                    final_min_g = final_min_g < image_data[ ( x + y * image_width ) * component_count + 1 ] ? final_min_g : image_data[ ( x + y * image_width ) * component_count + 1 ];
                    final_min_b = final_min_b < image_data[ ( x + y * image_width ) * component_count + 2 ] ? final_min_b : image_data[ ( x + y * image_width ) * component_count + 2 ];
                }
            }

            //write data to the correct memory address for the new image ( /2 )
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 0 ] = final_min_r;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 1 ] = final_min_g;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 2 ] = final_min_b;
            output_data[ ( ( pixelX / 2 ) + ( pixelY / 2 ) * ( image_width / 2 ) ) * component_count + 3 ] = image_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ];
        }
    }
}

//kernel for copying data from the output image data to the input image data
__global__ void copy_output_to_input( uint8_t *image_data, uint8_t *output_data, int image_width, int image_height, int component_count )
{
    int pixelX = threadIdx.x + blockIdx.x * 32;
    int pixelY = threadIdx.y + blockIdx.y * 32;

    //only execute if the current pixel x and y are within the boundery of the image
    if ( ( component_count == 4 ) && ( ( pixelX < image_width ) && ( pixelY < image_height ) ) )
    {
        image_data[ ( pixelX + pixelY * image_width ) * component_count + 0 ] = output_data[ ( pixelX + pixelY * image_width ) * component_count + 0 ];
        image_data[ ( pixelX + pixelY * image_width ) * component_count + 1 ] = output_data[ ( pixelX + pixelY * image_width ) * component_count + 1 ];
        image_data[ ( pixelX + pixelY * image_width ) * component_count + 2 ] = output_data[ ( pixelX + pixelY * image_width ) * component_count + 2 ];
        image_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ] = output_data[ ( pixelX + pixelY * image_width ) * component_count + 3 ];
    }
}

class png_image_data
{
    private:
        const int required_component_count = 4; //r g b a

        dim3 *blockSize;
        dim3 *gridSize;

        uint8_t *all_pixel_values;
        uint8_t *output_pixel_values;
        uint8_t *ptr_image_data_on_gpu;
        uint8_t *ptr_output_data_on_gpu;

        int image_width;
        int image_height;
        int used_component_count;
        int element_count;

        enum png_error_states
        {
            PNG_ERROR_STATE_NONE,
            PNG_ERROR_STATE_CANNOT_OPEN_PNG,
            PNG_ERROR_STATE_COMPONENT_COUNT
        } png_error_state;

    public:
        enum png_kernel_option_t
        {
            PNG_EDGE_DETECTION,
            PNG_GRAY_SCALE,
            PNG_AVERAGE_POOLING,
            PNG_MAX_POOLING,
            PNG_MIN_POOLING
        };
        std::vector< enum png_kernel_option_t > png_kernel_options;
        
        png_image_data( std::string filename )
        {
            std::cout << "loading .png file " << filename << "\n\r";
            this->all_pixel_values = stbi_load( filename.c_str(), &( this->image_width ), &( this->image_height ), &( this->used_component_count ), this->required_component_count );
            
            //make sure the image got read correctly
            if ( !( this->all_pixel_values ) )
            {
                std::cout << "error: failed to open file\r\n";
                this->png_error_state = PNG_ERROR_STATE_CANNOT_OPEN_PNG;
            }
            else if ( this->used_component_count != this->required_component_count )
            {
                std::cout << "error: wrong image format ( component count = " << this->used_component_count << " )\r\n";
                this->png_error_state = PNG_ERROR_STATE_COMPONENT_COUNT;
            }
            else
            {
                //if image got read correclty parse the newly given image value's into the class
                this->element_count = this->image_width * this->image_height * this->used_component_count;
                std::cout << "data read: " << std::to_string( this->element_count ) << " elements\r\n";
                
                this->output_pixel_values = new uint8_t[ this->element_count ];
                for ( int i = 0; i < this->element_count; ++i )
                {
                    this->output_pixel_values[i] = this->all_pixel_values[i];
                }

                //cuda kernel value's
                ptr_image_data_on_gpu = nullptr;
                ptr_output_data_on_gpu = nullptr;

                blockSize = new dim3( 32, 32 );
                gridSize = new dim3( ( image_width + 32 - ( image_width % 32 ) ) / this->blockSize->x, ( image_height + 32 - ( image_height % 32 ) ) / this->blockSize->y );

                this->png_error_state = PNG_ERROR_STATE_NONE;
            }
        }

        ~png_image_data()
        {
            stbi_image_free( this->all_pixel_values );
            delete this->all_pixel_values;
        }

        int get_last_error()
        {
            switch ( this->png_error_state )
            {
                case PNG_ERROR_STATE_CANNOT_OPEN_PNG:
                    return -11;
                case PNG_ERROR_STATE_COMPONENT_COUNT:
                    return -12;
                case PNG_ERROR_STATE_NONE:
                default:
                    return 0;
            };
        }

        //call edge detection kernel
        void do_edge_detection()
        {
            edge_detection_on_gpu<<<*( this->gridSize ), *( this->blockSize )>>>( this->ptr_image_data_on_gpu, this->ptr_output_data_on_gpu, this->image_width, this->image_height, this->used_component_count );
        }

        //call gray scale kernel
        void do_gray_scale()
        {
            gray_scale_on_gpu<<<*( this->gridSize ), *( this->blockSize )>>>( this->ptr_image_data_on_gpu, this->ptr_output_data_on_gpu, this->image_width, this->image_height, this->used_component_count );
        }

        //call average pooling kernel
        void do_average_pooling()
        {
            average_pooling_on_gpu<<<*( this->gridSize ), *( this->blockSize )>>>( this->ptr_image_data_on_gpu, this->ptr_output_data_on_gpu, this->image_width, this->image_height, this->used_component_count );
        }

        //call max pooling kernel
        void do_max_pooling()
        {
            max_pooling_on_gpu<<<*( this->gridSize ), *( this->blockSize )>>>( this->ptr_image_data_on_gpu, this->ptr_output_data_on_gpu, this->image_width, this->image_height, this->used_component_count );
        }

        //call min pooling kernel
        void do_min_pooling()
        {
            min_pooling_on_gpu<<<*( this->gridSize ), *( this->blockSize )>>>( this->ptr_image_data_on_gpu, this->ptr_output_data_on_gpu, this->image_width, this->image_height, this->used_component_count );
        }

        //call copy output kernel
        void do_copy_output_to_input()
        {
            copy_output_to_input<<<*( this->gridSize ), *( this->blockSize )>>>( this->ptr_image_data_on_gpu, this->ptr_output_data_on_gpu, this->image_width, this->image_height, this->used_component_count );
        }

        //run all the kernels in a single stream
        void run_kernels()
        {
            std::cout << "Running CUDA Kernel...\r\n";
            std::cout << "Copy data to GPU...\r\n";

            hipMalloc( &( this->ptr_image_data_on_gpu ), this->element_count );
            hipMalloc( &( this->ptr_output_data_on_gpu ), this->element_count );
            hipMemcpy( this->ptr_image_data_on_gpu, this->all_pixel_values, this->element_count, hipMemcpyHostToDevice );
            hipMemcpy( this->ptr_output_data_on_gpu, this->output_pixel_values, this->element_count, hipMemcpyHostToDevice );

            //loop over all kernel in the stream
            for( png_kernel_option_t kernel : this->png_kernel_options )
            {
                switch ( kernel )
                {
                    case PNG_EDGE_DETECTION:
                        do_edge_detection();
                        this->image_width -= 3;
                        this->image_height -= 3;
                        break;
                    case PNG_GRAY_SCALE:
                        do_gray_scale();
                        break;
                    case PNG_MAX_POOLING:
                        do_max_pooling();
                        this->image_width /= 2;
                        this->image_height /= 2;
                        break;
                    case PNG_MIN_POOLING:
                        do_min_pooling();
                        this->image_width /= 2;
                        this->image_height /= 2;
                        break;
                    case PNG_AVERAGE_POOLING:
                        do_average_pooling();
                        this->image_width /= 2;
                        this->image_height /= 2;
                        break;
                }

                do_copy_output_to_input();
            }

            hipDeviceSynchronize();

            std::cout << "Copy data from GPU...\r\n";

            hipMemcpy( this->all_pixel_values, this->ptr_image_data_on_gpu, this->element_count, hipMemcpyDeviceToHost );
            hipMemcpy( this->output_pixel_values, this->ptr_output_data_on_gpu, this->element_count, hipMemcpyDeviceToHost );
            hipFree( this->ptr_image_data_on_gpu );
            hipFree( this->ptr_output_data_on_gpu );

            std::cout << "done!\r\n";
        }

        //write the image to the given filename
        void write_to_output( std::string output_file_name )
        {
            std::cout << "writing png to disk...\r\n";
            stbi_write_png( output_file_name.c_str(), this->image_width, this->image_height, this->used_component_count, this->output_pixel_values, ( this->used_component_count * this->image_width ) );
        }

        void print_data()
        {
            std::cout << " imagesize: " << this->image_width << "x" << this->image_height << "\r\n" << " component count: " << this->used_component_count << "\r\n\r\n";
        }
};

int main( int argc, char *argv[] )
{
    //init check
    if ( argc < 3 )
    {
        std::cout << "error: not enough arguments, expected minimum two\n\r";
        return -1;
    }

    //init
    std::string png_file_filename( argv[1] );
    
    png_image_data *input_file = new png_image_data::png_image_data( png_file_filename );
    if ( input_file->get_last_error() != 0 ) return -1;
    
    std::cout << "loaded .png file\r\n";
    input_file->print_data();
    
    //fill the buffer for calling all the kernels
    for ( int i = 2; i < argc; ++i )
    {
        std::string operation( argv[i] );

        if ( operation == "gray_scale" ) input_file->png_kernel_options.push_back( png_image_data::PNG_GRAY_SCALE );
        else if ( operation == "edge_detection" ) input_file->png_kernel_options.push_back( png_image_data::PNG_EDGE_DETECTION );
        else if ( operation == "min_pooling" ) input_file->png_kernel_options.push_back( png_image_data::PNG_MIN_POOLING );
        else if ( operation == "max_pooling" ) input_file->png_kernel_options.push_back( png_image_data::PNG_MAX_POOLING );
        else if ( operation == "average_pooling" ) input_file->png_kernel_options.push_back( png_image_data::PNG_AVERAGE_POOLING );
        else std::cout << "error: invalid operation " << operation << "\n\r";
    }
    input_file->run_kernels();

    input_file->print_data();
    input_file->write_to_output( "output_file.png" );

    //cleanup
    std::cout << "DONE\r\n";
    return 0;
}

